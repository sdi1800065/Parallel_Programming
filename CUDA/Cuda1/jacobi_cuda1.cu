#include "hip/hip_runtime.h"
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "timestamp.h"

#define N 32 // 2-4-8-16-32 max 32
#define FRACTION_CEILING(numerator, denominator) ((numerator+denominator-1)/denominator)
#define SRC(XX,YY) src[(YY)*maxXCount+(XX)]
#define DST(XX,YY) dest[(YY)*maxXCount+(XX)]


__global__ void one_jacobi_itteration(int maxXCount, int maxYCount,double*dest,double *src,double deltaX, double deltaY,double cc,double cx,double cy,double relax,double alpha,double *loop_error_c)
{
    __shared__ double  temp[N*N]; 
    int index = blockDim.x * threadIdx.y + threadIdx.x;
    temp[index]=0.0;
    //Calculate the column index of the Pd element, denote by x
     int x = threadIdx.x + blockIdx.x * blockDim.x+1; 
    //Calculate the row index of the Pd element, denote by y
    int y = threadIdx.y + blockIdx.y * blockDim.y+1; 
    
    if (x < maxXCount-1 && y < maxYCount -1) {
  	    double fY =  -1.0 + (y-1)*deltaY;
       	double fX = -1.0 + (x-1)*deltaX;
        double updateVal = ((SRC(x-1,y) + SRC(x+1,y))*cx + (SRC(x,y-1) + SRC(x,y+1))*cy + SRC(x,y)*cc - ((fX*fX-1.0)*(alpha*(1.0-fY*fY) + 2.0) - 2.0*(1.0 - fY*fY)))/cc;
		DST(x,y) = SRC(x,y) - relax*updateVal;	
		temp[index] = updateVal*updateVal;
        
        // Synchronize (ensure all the data is available) 
        __syncthreads();
        
        //apply stencil

        for(int offset=N*N/2;offset>=1;offset=offset/2)
        {
            if(index<offset)
            {
                temp[index]=temp[index]+temp[index+offset];
            }
        __syncthreads();
        }
        if(threadIdx.x + threadIdx.y == 0) {
            atomicAdd(loop_error_c, temp[0]);
        }
    }
}

/**********************************************************
 * Checks the error between numerical and exact solutions
 **********************************************************/
static inline double checkSolution(double xStart, double yStart,
                     int maxXCount, int maxYCount,
                     double *u,
                     double deltaX, double deltaY,
                     double alpha)
{
#define U(XX,YY) u[(XX)*maxYCount+(YY)]
    int x, y;
    double fX, fY;
    double localError, error = 0.0;

    for (y = 1; y < (maxYCount-1); y++)
    {
        fY = yStart + (y-1)*deltaY;
        for (x = 1; x < (maxXCount-1); x++)
        {
            fX = xStart + (x-1)*deltaX;
            localError = U(x,y) - (1.0-fX*fX)*(1.0-fY*fY);
            error += localError*localError;
        }
    }
	return error;
}


int main(int argc, char **argv)
{
    int n, m, mits;
    double alpha, tol, relax;
    double error;
    double *u, *u_old, *loop_error, *tmp,*u1;
    int allocCount;
    int iterationCount;
    timestamp t1;
    double t2;

//    printf("Input n,m - grid dimension in x,y direction:\n");
    scanf("%d,%d", &n, &m);
//    printf("Input alpha - Helmholtz constant:\n");
    scanf("%lf", &alpha);
//    printf("Input relax - successive over-relaxation parameter:\n");
    scanf("%lf", &relax);
//    printf("Input tol - error tolerance for the iterrative solver:\n");
    scanf("%lf", &tol);
//    printf("Input mits - maximum solver iterations:\n");
    scanf("%d", &mits);

    printf("-> %d, %d, %g, %g, %g, %d\n", n, m, alpha, relax, tol, mits);

    allocCount = (n+2)*(m+2);
    
    // Those three calls also zero the boundary elements
    u1 = 	(double*)calloc(allocCount, sizeof(double));

    hipMalloc((void**)&u_old, allocCount * sizeof(double));
    hipMalloc((void**)&u, allocCount * sizeof(double));
    hipMalloc((void**)&loop_error, 1 * sizeof(double));


    hipMemset(u, 0, allocCount * sizeof(double));
    hipMemset(u_old, 0, allocCount * sizeof(double));
    hipMemset(loop_error, 0, sizeof(double));

    // Solve in [-1, 1] x [-1, 1]
    double xLeft = -1.0, xRight = 1.0;
    double yBottom = -1.0, yUp = 1.0;

    double deltaX = (xRight-xLeft)/(n-1);
    double deltaY = (yUp-yBottom)/(m-1);
	
    // Coefficients
    double cx = 1.0/(deltaX*deltaX);
    double cy = 1.0/(deltaY*deltaY);
    double cc = -2.0*cx-2.0*cy-alpha;
    
    iterationCount = 0;
    error = HUGE_VAL;


    dim3 dimBl(N, N);
    dim3 dimGr(FRACTION_CEILING(n+2, N), FRACTION_CEILING(m+2, N));

    clock_t start , diff;
    start = clock();
    t1 = getTimestamp();
    

    /* Iterate as long as it takes to meet the convergence criterion */
    while (iterationCount < mits && error > tol)
    {
        hipMemset(loop_error, 0, sizeof(double));
        
        one_jacobi_itteration<<<dimGr,dimBl>>>((m+2),(n+2),u,u_old,deltaX,deltaY,cc,cx,cy,relax,alpha,loop_error);
                
        hipMemcpy(&error,loop_error,sizeof(double),hipMemcpyDeviceToHost);

        iterationCount++;
        // Swap the buffers
        tmp = u_old;
        u_old = u;
        u = tmp;
        error = sqrt(error)/(n*m);
    }

	t2 = getElapsedtime(t1) / 1000.0;

    // u1 holds the solution after the most recent buffers swap
	hipMemcpy(u1,u_old,sizeof(double)*allocCount,hipMemcpyDeviceToHost);
	
    printf( "Iterations=%3d Elapsed MPI Wall time is %f\n", iterationCount,t2);
	
    
	diff = clock() - start;
	double sec = diff/(double) CLOCKS_PER_SEC;
	printf("Clock Time %lf \n",sec);
	printf("Residual %g\n",error);
	printf("The sum error of the iterative solution is is %g\n", sqrt(checkSolution(-1.0,-1.0, n+2, m+2, u1, deltaX, deltaY, alpha))/(m*n));
    
    return 0;
}
